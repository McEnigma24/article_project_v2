#include "__preprocessor__.h" // to use my core lib i had to -> mv .cpp .cu --- so when downloading lib first time you have to run ./start 2 time first with -c that solo
#include "openMP_test.h"
#include "Multi_Dimension_View_Array.hpp"
#include "fstream"
#include "parallel_common.h"

#define sphere_radius ( 1.0f )
#define ID_RANGE ( 5 )

typedef float unit;
// typedef double unit;

struct ovito_XYZ_format_obj
{
    int id;
    unit x, y, z;

    GPU_LINE(__device__ __host__)
    void init(int id, unit x, unit y, unit z)
    {
        this->id = id;
        this->x = x * 2.4f;
        this->y = y * 2.4f;
        this->z = z * 2.4f;
    }

    static void dump_to_file(const ovito_XYZ_format_obj* arr, size_t size)
    {
        CPU_LINE(const string& FILEPATH = "output/cpu.xyz";)
        GPU_LINE(const string& FILEPATH = "output/gpu.xyz";)

        std::ofstream fout(FILEPATH, std::ios::out | std::ios::app);
        if (!fout)
        {
            std::cerr << "Nie można otworzyć pliku do nadpisania: " << FILEPATH << "\n";
            return;
        }

        fout << size << endl;
        fout << " " << endl;
        for(int i=0; i<size; i++)
        {
            const auto& obj = arr[i];
            fout << obj.x << " " << obj.y << " " << obj.z << " " << obj.id << endl;
        }

        if (!fout)
        {
            std::cerr << "Błąd podczas zapisu do pliku: " << FILEPATH << "\n";
            fout.close();
        }
        fout.close();
    }
};

class Sphere : public ovito_XYZ_format_obj
{
public:
    static void dump_to_file(const Multi_Dimension_View_Array<Sphere>& arr)
    {
        ovito_XYZ_format_obj::dump_to_file(arr.get_vector().data(), arr.get_vector().size());
    }
};

void initialize_sim(Multi_Dimension_View_Array<Sphere>& arr)
{
    for(int z=0; z<arr.get_depth(); z++)
        for(int y=0; y<arr.get_height(); y++)
            for(int x=0; x<arr.get_width(); x++)
                arr.get(x, y, z)->init((rand() % ID_RANGE), x, y, z);
}

template<typename T, size_t N>
class ObjTracker
{
    T arr_of_objects[N];
    size_t current_array_index;

    size_t next_value()
    {
        return (current_array_index + 1) % N;
    }

public:

    ObjTracker(i64 width, i64 height, i64 depth)
        : current_array_index(0)
    {
        for(size_t i=0; i<N; i++)
        {
            arr_of_objects[i].set_sizes(width, height, depth);
            initialize_sim(arr_of_objects[i]);
        }
    }

    T& get_current_obj()
    {
        return arr_of_objects[current_array_index];
    }

    T& get_next_obj()
    {
        return arr_of_objects[next_value()];
    }

    void next_cycle()
    {
        current_array_index = next_value();
    }

    void reset_to_start()
    {
        current_array_index = 0;
    }
};

template<size_t N>
void per_sphere(ObjTracker<Multi_Dimension_View_Array<Sphere>, N>& obj_tracker, const coords& my_coords)
{
    constexpr int neighbor_range = 1;

    auto& current_array = obj_tracker.get_current_obj();
    auto& next_arr = obj_tracker.get_next_obj();

    const int my_id = current_array.get(my_coords.x, my_coords.y, my_coords.z)->id;
    int& my_next_id = next_arr.get(my_coords.x, my_coords.y, my_coords.z)->id;

    std::array<u8, ID_RANGE> id_tab;
    id_tab.fill(0);
    
    // pętla po sąsiadach
    for(int dz=-neighbor_range; dz<=neighbor_range; dz++)
        for(int dy=-neighbor_range; dy<=neighbor_range; dy++)
            for(int dx=-neighbor_range; dx<=neighbor_range; dx++)
            {
                if(dx == 0 && dy == 0 && dz == 0) continue;

                int nx = ((my_coords.x + dx) + current_array.get_width())     % current_array.get_width();
                int ny = ((my_coords.y + dy) + current_array.get_height())    % current_array.get_height();
                int nz = ((my_coords.z + dz) + current_array.get_depth())     % current_array.get_depth();

                const Sphere* neighbor = current_array.get(nx, ny, nz);
                
                id_tab[neighbor->id]++;
            }

    u8 most_frequent_id_index = 0;
    for(int i=1; i<ID_RANGE; i++)
    {
        if(id_tab[most_frequent_id_index] < id_tab[i])
        {
            most_frequent_id_index = i;
        }
    }

    my_next_id = most_frequent_id_index;
}

template<size_t N>
void dump_all_saved_states_to_file(ObjTracker<Multi_Dimension_View_Array<Sphere>, N>& obj_tracker)
{
    obj_tracker.reset_to_start();

    for(size_t i=0; i<N; i++)
    {
        Sphere::dump_to_file(obj_tracker.get_current_obj());
        obj_tracker.next_cycle();
    }
}

#ifdef BUILD_EXECUTABLE
int main(int argc, char* argv[])
{
    srand(time(NULL));
    constexpr int cube_side = 90;
    constexpr int sim_steps = 5;

    // ObjTracker - mogę mu podać tyle samo co sim_steps, wtedy zapisze jak już będzie po wszystkim -> albo batch, po którym zapiszę wszystko do pliku

    ObjTracker<Multi_Dimension_View_Array<Sphere>, sim_steps> obj_tracker(cube_side, cube_side, cube_side);

    time_stamp_reset();
    // #pragma omp parallel
    {
        for(int step = 0; step < (sim_steps - 1); step++)
        {
            auto& arr = obj_tracker.get_current_obj();
            
            // #pragma omp for schedule(static)
            for(int i=0; i<arr.get_total_number(); i++)
            {
                per_sphere(obj_tracker, arr.get_coords(i));
            }

            // #pragma omp barrier
            // #pragma omp single
            {
                obj_tracker.next_cycle();
            }
        }
    }
    time_stamp("FINISH");
    
    dump_all_saved_states_to_file(obj_tracker);

    time_stamp("io DONE");


    // zrobimy zwykły automat komórkowy montecarlo tylko 3D - w wrzucimy w Avitoo jako film .itd

    // potem możemy przemyśleć to przesunięcie i takie okrągłe sąsiedztwo (w sumie to będzie to samo, tylko przesunięte o ileś tam)


    return 0;
}
#endif